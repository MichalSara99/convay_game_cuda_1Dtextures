#include "hip/hip_runtime.h"
#include"convay_game.h"

#include<hip/hip_runtime.h>
#include<>
#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include<stdlib.h>
#include<time.h>


#define THREADS_PER_BLOCK 10

struct neighbours {
	int leftIdx;
	int topIdx;
	int topLeftIdx;
	int topRightIdx;
	int rightIdx;
	int bottomIdx;
	int bottomLeftIdx;
	int bottomRightIdx;
};

// global texture references:

texture<int> boardTex;
texture<int> roBoardTex;

__device__
int getCount(bool readOnly,int tid, int width, long size, neighbours n) {

	int cnt{ 0 };
	if (readOnly) {
		// check the corners and boundaries:
		if (tid == 0) {
			cnt += tex1Dfetch(roBoardTex, n.rightIdx) + tex1Dfetch(roBoardTex,n.bottomIdx) + tex1Dfetch(roBoardTex,n.bottomRightIdx);
			return cnt;
		}

		// bottom left corner:
		if (tid == (size - width)) {
			cnt += tex1Dfetch(roBoardTex,n.topIdx) + tex1Dfetch(roBoardTex,n.topRightIdx) + tex1Dfetch(roBoardTex,n.rightIdx);
			return cnt;
		}
		//	top right corner:
		if (tid == (width - 1)) {
			cnt += tex1Dfetch(roBoardTex,n.leftIdx) + tex1Dfetch(roBoardTex,n.bottomLeftIdx) + tex1Dfetch(roBoardTex,n.bottomIdx);
			return cnt;
		}
		//	bottom right corner:
		if (tid == (size - 1)) {
			cnt += tex1Dfetch(roBoardTex,n.topIdx) + tex1Dfetch(roBoardTex,n.topLeftIdx) + tex1Dfetch(roBoardTex,n.leftIdx);
			return cnt;
		}
		// left boundary of the board:
		if ((tid % width) == 0) {
			cnt += tex1Dfetch(roBoardTex,n.topIdx) + tex1Dfetch(roBoardTex,n.topRightIdx)
				+ tex1Dfetch(roBoardTex,n.rightIdx) + tex1Dfetch(roBoardTex,n.bottomRightIdx) + tex1Dfetch(roBoardTex,n.bottomIdx);
			return cnt;
		}
		// top boundary of the board:
		if ((tid >= 0) && (tid <= (width - 1))) {
			cnt += tex1Dfetch(roBoardTex, n.leftIdx) + tex1Dfetch(roBoardTex, n.bottomLeftIdx)
				+ tex1Dfetch(roBoardTex, n.bottomIdx) + tex1Dfetch(roBoardTex, n.bottomRightIdx) + tex1Dfetch(roBoardTex, n.rightIdx);
			return cnt;
		}
		// right boundary of the board:
		if (((tid + 1) % width) == 0) {
			cnt += tex1Dfetch(roBoardTex,n.topIdx) + tex1Dfetch(roBoardTex,n.topLeftIdx)
				+ tex1Dfetch(roBoardTex,n.leftIdx) + tex1Dfetch(roBoardTex,n.bottomLeftIdx) + tex1Dfetch(roBoardTex,n.bottomIdx);
			return cnt;
		}
		// bottom boundary of the board:
		if ((tid >= (size - width)) && (tid <= (size - 1))) {
			cnt += tex1Dfetch(roBoardTex,n.leftIdx) + tex1Dfetch(roBoardTex,n.topLeftIdx)
				+ tex1Dfetch(roBoardTex,n.topIdx) + tex1Dfetch(roBoardTex,n.topRightIdx) + tex1Dfetch(roBoardTex,n.rightIdx);
			return cnt;
		}

		cnt += tex1Dfetch(roBoardTex,n.leftIdx) + tex1Dfetch(roBoardTex,n.topLeftIdx) + tex1Dfetch(roBoardTex,n.topIdx) + 
			tex1Dfetch(roBoardTex,n.topRightIdx) + tex1Dfetch(roBoardTex,n.rightIdx) + tex1Dfetch(roBoardTex,n.bottomRightIdx) + 
			tex1Dfetch(roBoardTex,n.bottomIdx) + tex1Dfetch(roBoardTex,n.bottomLeftIdx);
		return cnt;
	}
	else {

		// check the corners and boundaries:
		if (tid == 0) {
			cnt += tex1Dfetch(boardTex,n.rightIdx) + tex1Dfetch(boardTex,n.bottomIdx) + tex1Dfetch(boardTex,n.bottomRightIdx);
			return cnt;
		}

		// bottom left corner:
		if (tid == (size - width)) {
			cnt += tex1Dfetch(boardTex,n.topIdx) + tex1Dfetch(boardTex,n.topRightIdx) + tex1Dfetch(boardTex,n.rightIdx);
			return cnt;
		}
		//	top right corner:
		if (tid == (width - 1)) {
			cnt += tex1Dfetch(boardTex,n.leftIdx) + tex1Dfetch(boardTex,n.bottomLeftIdx) + tex1Dfetch(boardTex,n.bottomIdx);
			return cnt;
		}
		//	bottom right corner:
		if (tid == (size - 1)) {
			cnt += tex1Dfetch(boardTex,n.topIdx) + tex1Dfetch(boardTex,n.topLeftIdx) + tex1Dfetch(boardTex,n.leftIdx);
			return cnt;
		}
		// left boundary of the board:
		if ((tid % width) == 0) {
			cnt += tex1Dfetch(boardTex,n.topIdx) + tex1Dfetch(boardTex,n.topRightIdx)
				+ tex1Dfetch(boardTex,n.rightIdx) + tex1Dfetch(boardTex,n.bottomRightIdx) + tex1Dfetch(boardTex,n.bottomIdx);
			return cnt;
		}
		// top boundary of the board:
		if ((tid >= 0) && (tid <= (width - 1))) {
			cnt += tex1Dfetch(boardTex,n.leftIdx) + tex1Dfetch(boardTex,n.bottomLeftIdx)
				+ tex1Dfetch(boardTex,n.bottomIdx) + tex1Dfetch(boardTex,n.bottomRightIdx) + tex1Dfetch(boardTex,n.rightIdx);
			return cnt;
		}
		// right boundary of the board:
		if (((tid + 1) % width) == 0) {
			cnt += tex1Dfetch(boardTex,n.topIdx) + tex1Dfetch(boardTex,n.topLeftIdx)
				+ tex1Dfetch(boardTex,n.leftIdx) + tex1Dfetch(boardTex,n.bottomLeftIdx) + tex1Dfetch(boardTex,n.bottomIdx);
			return cnt;
		}
		// bottom boundary of the board:
		if ((tid >= (size - width)) && (tid <= (size - 1))) {
			cnt += tex1Dfetch(boardTex,n.leftIdx) + tex1Dfetch(boardTex,n.topLeftIdx)
				+ tex1Dfetch(boardTex,n.topIdx) + tex1Dfetch(boardTex,n.topRightIdx) + tex1Dfetch(boardTex,n.rightIdx);
			return cnt;
		}

		cnt += tex1Dfetch(boardTex,n.leftIdx) + tex1Dfetch(boardTex,n.topLeftIdx) + tex1Dfetch(boardTex,n.topIdx) + 
			tex1Dfetch(boardTex,n.topRightIdx) + tex1Dfetch(boardTex,n.rightIdx) + tex1Dfetch(boardTex,n.bottomRightIdx) + 
			tex1Dfetch(boardTex,n.bottomIdx)+ tex1Dfetch(boardTex,n.bottomLeftIdx);
		return cnt;
	}
	
}

__global__
void convay_kernel(int *board,bool readOnly, long long size) {
	unsigned int const tidx = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int const tidy = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int const tid = tidx + tidy * gridDim.x * blockDim.x;

	if (tid >= size)return;

	// width of the board:
	int const width = gridDim.x * blockDim.x;

	// compute indices of all neighbors:
	neighbours n;
	n.leftIdx = tid - 1;
	n.topIdx = tid - width;
	n.topLeftIdx = n.topIdx - 1;
	n.topRightIdx = n.topIdx + 1;
	n.rightIdx = tid + 1;
	n.bottomIdx = tid + width;
	n.bottomLeftIdx = n.bottomIdx - 1;
	n.bottomRightIdx = n.bottomIdx + 1;

	int cnt = getCount(readOnly, tid, width, size, n);
	int check{ 0 };

	if (readOnly) {
		check = tex1Dfetch(roBoardTex, tid);
	}
	else {
		check = tex1Dfetch(boardTex, tid);
	}

	if (check == 0)
		board[tid] = (int)(cnt == 3);
	else
		board[tid] = (int)((cnt == 2) || (cnt == 3));


}


__global__
void init_random(unsigned int seed, hiprandState_t* states, long size) {
	unsigned int const tidx = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int const tidy = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int const tid = tidx + tidy * gridDim.x * blockDim.x;
	if (tid >= size)return;

	hiprand_init(seed, tid, 0, &states[tid]);
}

__global__
void generateBoard(int* board, hiprandState_t* states, long size) {
	unsigned int const tidx = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int const tidy = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int const tid = tidx + tidy * gridDim.x * blockDim.x;

	if (tid >= size)return;

	board[tid] = hiprand(&states[tid]) % 2;

}


void ConvayGame::print(int* board, long counter) {
	std::size_t const totalSize = boardSize_ * boardSize_;
	std::cout << counter << ".generation\n| ";
	for (std::size_t t = 0; t < totalSize; ++t) {
		if ((t > 0) && (t % boardSize_) == 0)
			std::cout << "|\n| ";
		std::cout << board[t] << " ";
	}
	std::cout << "|\n";
}


void ConvayGame::initBoard(int* DBoard) {
	dim3 const blockSize(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
	dim3 const gridSize = dim3((boardSize_ + blockSize.x - 1) / blockSize.x,
		(boardSize_ + blockSize.y - 1) / blockSize.y);
	long const totalSize = boardSize_ * boardSize_;

	hiprandState_t* states;
	hipMalloc((void**)&states, sizeof(hiprandState_t) * totalSize);
	init_random << <gridSize, blockSize >> > (time(0), states, totalSize);
	generateBoard << <gridSize, blockSize >> > (DBoard, states, totalSize);
	hipFree(states);
}

void ConvayGame::singleGeneration(int* board,bool readOnly) {
	dim3 const blockSize(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
	dim3 const gridSize = dim3((boardSize_ + blockSize.x - 1) / blockSize.x,
		(boardSize_ + blockSize.y - 1) / blockSize.y);
	long const totalSize = boardSize_ * boardSize_;
	convay_kernel << <gridSize, blockSize >> > (board, readOnly, totalSize);
}



void ConvayGame::play() {
	int const totalSize = boardSize_ * boardSize_;

	int* hBoard = (int*)malloc(sizeof(int) * totalSize);
	int* dBoard;
	int* dROBoard;
	hipMalloc((void**)&dBoard, sizeof(int) * totalSize);
	hipMalloc((void**)&dROBoard, sizeof(int) * totalSize);

	hipBindTexture(NULL, roBoardTex, dROBoard, totalSize);
	hipBindTexture(NULL, boardTex, dBoard, totalSize);


	initBoard(dROBoard);
	hipMemcpy(hBoard, dROBoard, sizeof(int) * totalSize,
		hipMemcpyKind::hipMemcpyDeviceToHost);
	print(hBoard, 0);
	volatile bool readOnly = true;
	for (std::size_t t = 1; t < generations_; ++t) {
		int* in, * out;
		if (readOnly) {
			in = dROBoard;
			out = dBoard;
		}
		else {
			in = dBoard;
			out = dROBoard;
		}
		singleGeneration(out, readOnly);
		hipMemcpy(hBoard, dBoard, sizeof(int) * totalSize,
			hipMemcpyKind::hipMemcpyDeviceToHost);
		print(hBoard, t);
		readOnly != readOnly;
	}


	free(hBoard);
	hipUnbindTexture(roBoardTex);
	hipUnbindTexture(boardTex);
	hipFree(dBoard);
	hipFree(dROBoard);

}